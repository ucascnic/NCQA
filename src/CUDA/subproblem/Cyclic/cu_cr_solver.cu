#include "cu_cr_solver.h"
#include "cu_cr_internal.h"
#include <cu_cr_functors.cu>

#include <hip/hip_runtime.h>
#include <math.h>
#include <thread>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/system/cuda/execution_policy.h>

/*
* For method documentation see cu_cr_internal.h unless otherwise specified.
*/

namespace cyclic_reduction{

void Solve(int size, double *d_vect_a,
               double * d_vect_b,
               double * d_vect_c,
               double * d_vect_d,
               double * d_vect_results){

    thrust::device_vector<double> d_vect_x(size,0.00),
		d_vect_a_prime(size,0.00),
		d_vect_c_prime(size,0.00),
		d_vect_temp(size,0.00);


//Define and create Cuda Streams
	
	hipStream_t s1,s2,s3;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	hipStreamCreate(&s3);

//Foward Reduction Phase

	int level = 1;
	while(level < size){

	//AlphaBeta Methods
		d_vect_x = d_vect_d;

		LowerAlphaBeta(size,level,
             thrust::device_pointer_cast(d_vect_a),
             d_vect_a_prime.data(),
             thrust::device_pointer_cast(d_vect_b)
		);

		UpperAlphaBeta(size, level,
             thrust::device_pointer_cast(d_vect_b),
             thrust::device_pointer_cast(d_vect_c),
			d_vect_c_prime.data()
		);
	
	
	//Front Methods
		
		MainFront(size, level,
			d_vect_a_prime.data(),
             thrust::device_pointer_cast(d_vect_b),
             thrust::device_pointer_cast(d_vect_c),
			d_vect_temp.data()
		);

		SolutionFront(size, level,
			d_vect_a_prime.data(),
            thrust::device_pointer_cast(d_vect_d),
			d_vect_x.data(),
			d_vect_temp.data()
		);

		LowerFront(size, level,
             thrust::device_pointer_cast(d_vect_a),
			d_vect_a_prime.data()
		);

	//Back Methods

		MainBack(size, level,
             thrust::device_pointer_cast(d_vect_a),
			d_vect_c_prime.data(),
             thrust::device_pointer_cast(d_vect_b),
			d_vect_temp.data()
		);

		SolutionBack(size, level,
			d_vect_c_prime.data(),
            thrust::device_pointer_cast(d_vect_d),
			d_vect_x.data(),
			d_vect_temp.data()
		);

		UpperBack(size, level,
             thrust::device_pointer_cast(d_vect_c),
             d_vect_c_prime.data()
		);			

	//Set up diagonals for next reduction level
        //d_vect_a = d_vect_a_prime;
        hipMemcpy(d_vect_a,thrust::raw_pointer_cast(d_vect_a_prime.data()),
                   sizeof(double)*size,hipMemcpyDeviceToDevice);
        hipMemcpy(d_vect_c,thrust::raw_pointer_cast(d_vect_c_prime.data()),
                   sizeof(double)*size,hipMemcpyDeviceToDevice);
        //d_vect_c = d_vect_c_prime;
        //d_vect_d = d_vect_x;
        hipMemcpy(d_vect_d,thrust::raw_pointer_cast(d_vect_x.data()),
                   sizeof(double)*size,hipMemcpyDeviceToDevice);
		level *= 2;
	}

//Backward Substitution Phase

		thrust::transform(
            d_vect_d.begin(), d_vect_d.end(),
            d_vect_b.begin(),
            d_vect_results.begin(),
			thrust::divides<double>()
		);



	hipStreamDestroy(s1);
	hipStreamDestroy(s2);
	hipStreamDestroy(s3);	
				
    return ;
}


void LowerAlphaBeta(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime, DPtrD d_ptr_b){

	thrust::transform(
		d_ptr_a + level, d_ptr_a + n,
		d_ptr_b,
		d_ptr_a_prime + level,
		AlphaBeta()
	);
		
}

void UpperAlphaBeta(int n, int level, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	thrust::transform(
		d_ptr_c , d_ptr_c + (n-level),
		d_ptr_b + level,
		d_ptr_c_prime,
		AlphaBeta()
	);

}

void MainFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_temp){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_c,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b + level, d_ptr_b + n,
		d_ptr_temp,
		d_ptr_b + level,
		thrust::plus<double>()
	);

}

void SolutionFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_d, DPtrD d_ptr_x, DPtrD d_ptr_temp ){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_d,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x + level, d_ptr_x + n,
		d_ptr_temp,
		d_ptr_x + level,
		thrust::plus<double>()
	);

}

void LowerFront(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_a,
		d_ptr_a_prime + level,
		thrust::multiplies<double>()
	);	
}

void MainBack(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_c_prime, DPtrD d_ptr_b, DPtrD d_ptr_temp){

	thrust::transform(
		d_ptr_c_prime , d_ptr_c_prime + (n - level),
		d_ptr_a + level,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b , d_ptr_b + (n - level),
		d_ptr_temp,
		d_ptr_b,
		thrust::plus<double>()
	);
}

void SolutionBack(int n, int level, DPtrD d_ptr_c_prime, DPtrD d_ptr_d, DPtrD d_ptr_x, DPtrD d_ptr_temp){
	
	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_d + level,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x , d_ptr_x + (n-level),
		d_ptr_temp,
		d_ptr_x,
		thrust::plus<double>()
	);

}


void UpperBack(int n, int level, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_c + level,
		d_ptr_c_prime,
		thrust::multiplies<double>()
	);	
}

}//END - namespace
