#include "hip/hip_runtime.h"
#include <cstdio>
#include "PCR_Device_functions.h"

__global__ void list_print(int nmax, float * in) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    printf("Thread %i shows %f \n", i, in[i]);
}


__global__ void Solve_Kernel(
    float * alist, float * blist, float * clist, float * dlist, float * xlist,
    int iter_max, int DMax) {

    int idx_row = blockIdx.x*blockDim.x + threadIdx.x;
    int row_max = DMax - 1;

    int stride = 1;
    int next_stride = stride;

    float a1, b1, c1, d1;
    float k01, k21, c01, a21, d01, d21;

    bool next_or_ot = true;
    int accum;

    for (int iter = 0; iter < iter_max; iter++) {

        if ( next_or_ot ) {

            next_stride = stride<<1;

            // 1    for updating 'a'
            if ((idx_row - stride)<0) {
            // 1.1  if it is the 'first' line
                a1 = 0.0f;
                k01 = 0.0f;
                c01 = 0.0f;
                d01 = 0.0f;
            } else if ((idx_row - next_stride)<0) {
            // 1.2  if no place for 'a'
                a1 = 0.0f;
                k01 = alist[idx_row]/blist[idx_row - stride];
                c01 = clist[idx_row - stride]*k01;
                d01 = dlist[idx_row - stride]*k01;
            } else {
            // 1.3  for rest general rows
                k01 = alist[idx_row]/blist[idx_row - stride];
                a1 = -alist[idx_row - stride]*k01;
                c01 = clist[idx_row - stride]*k01;
                d01 = dlist[idx_row - stride]*k01;
            }

            // 2    for updating 'c'
            if ((idx_row + stride)>row_max) {
            // 2.1  if it is the 'last' line
                c1 = 0.0f;
                k21 = 0.0f;
                a21 = 0.0f;
                d21 = 0.0f;
            } else if ((idx_row + next_stride)>row_max) {
                c1 = 0.0f;
                k21 = clist[idx_row]/blist[idx_row + stride];
                a21 = alist[idx_row + stride]*k21;
                d21 = dlist[idx_row + stride]*k21;
            } else {
                k21 = clist[idx_row]/blist[idx_row + stride];
                c1 = -clist[idx_row + stride]*k21;
                a21 = alist[idx_row + stride]*k21;
                d21 = dlist[idx_row + stride]*k21;
            }
            // 3   for updating 'b'
            b1 = blist[idx_row] - c01 - a21;
            // 4   for updating 'd'
            d1 = dlist[idx_row] - d01 - d21;

            stride = next_stride;

            //Determine if this line has reached the bi-set
            int pos = idx_row-2*stride;
            accum = 0;
            for ( size_t iter = 0; iter<5; iter++ ) {
                if (pos >=0 && pos < DMax) accum++;
                pos+=stride;
            }
            if (accum < 3) {
                next_or_ot = false;//Turn of for ever
            }

        }

        __syncthreads();
        __threadfence();
        alist[idx_row] = a1;
        blist[idx_row] = b1;
        clist[idx_row] = c1;
        dlist[idx_row] = d1;

    }

    if ( accum==1 ) {
        xlist[idx_row] = dlist[idx_row] / blist[idx_row];
    } else if ( (idx_row-stride)<0 ) {
        int i = idx_row; int k = idx_row+stride;
        float f = clist[i]/blist[k];
        xlist[i] = (dlist[i]-dlist[k]*f)/(blist[i]-alist[k]*f);
    } else {
        int i = idx_row - stride; int k = idx_row;
        float f = alist[k]/blist[i];
        xlist[k] = (dlist[k]-dlist[i]*f)/(blist[k]-clist[i]*f);
    }

}


__global__ void Solve_KernelD(
    double * alist, double * blist, double * clist, double * dlist, double * xlist,
    int iter_max, int DMax) {

    int idx_row = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx_row  >= DMax)
        return ;

    int row_max = DMax - 1;

    int stride = 1;
    int next_stride = stride;

    double a1, b1, c1, d1;
    double k01, k21, c01, a21, d01, d21;

    bool next_or_ot = true;
    int accum;

    for (int iter = 0; iter < iter_max; iter++) {

        if ( next_or_ot ) {

            next_stride = stride<<1;

            // 1    for updating 'a'
            if ((idx_row - stride)<0) {
            // 1.1  if it is the 'first' line
                a1 = 0.0;
                k01 = 0.0;
                c01 = 0.0;
                d01 = 0.0;
            } else if ((idx_row - next_stride)<0) {
            // 1.2  if no place for 'a'
                a1 = 0.0;
                k01 = alist[idx_row]/blist[idx_row - stride];
                c01 = clist[idx_row - stride]*k01;
                d01 = dlist[idx_row - stride]*k01;
            } else {
            // 1.3  for rest general rows
                k01 = alist[idx_row]/blist[idx_row - stride];
                a1 = -alist[idx_row - stride]*k01;
                c01 = clist[idx_row - stride]*k01;
                d01 = dlist[idx_row - stride]*k01;
            }

            // 2    for updating 'c'
            if ((idx_row + stride)>row_max) {
            // 2.1  if it is the 'last' line
                c1 = 0.0;
                k21 = 0.0;
                a21 = 0.0;
                d21 = 0.0;
            } else if ((idx_row + next_stride)>row_max) {
                c1 = 0.0;
                k21 = clist[idx_row]/blist[idx_row + stride];
                a21 = alist[idx_row + stride]*k21;
                d21 = dlist[idx_row + stride]*k21;
            } else {
                k21 = clist[idx_row]/blist[idx_row + stride];
                c1 = -clist[idx_row + stride]*k21;
                a21 = alist[idx_row + stride]*k21;
                d21 = dlist[idx_row + stride]*k21;
            }
            // 3   for updating 'b'
            b1 = blist[idx_row] - c01 - a21;
            // 4   for updating 'd'
            d1 = dlist[idx_row] - d01 - d21;

            stride = next_stride;

            //Determine if this line has reached the bi-set
            int pos = idx_row-2*stride;
            accum = 0;
            for ( size_t iter = 0; iter<5; iter++ ) {
                if (pos >=0 && pos < DMax) accum++;
                pos+=stride;
            }
            if (accum < 3) {
                next_or_ot = false;//Turn of for ever
            }

        }

        __syncthreads();__threadfence();

        alist[idx_row] = a1;
        blist[idx_row] = b1;
        clist[idx_row] = c1;
        dlist[idx_row] = d1;

    }

    if ( accum==1 ) {
        xlist[idx_row] = dlist[idx_row] / blist[idx_row];
    } else if ( (idx_row-stride)<0 ) {
        int i = idx_row; int k = idx_row+stride;
        double f = clist[i]/blist[k];
        xlist[i] = (dlist[i]-dlist[k]*f)/(blist[i]-alist[k]*f);
    } else {
        int i = idx_row - stride; int k = idx_row;
        double f = alist[k]/blist[i];
        xlist[k] = (dlist[k]-dlist[i]*f)/(blist[k]-clist[i]*f);
    }

}

